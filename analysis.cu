#include "hip/hip_runtime.h"
#include "analysis.h"
#include "timer.h"

#include <iostream>
#include <cstdlib>

AnalysisResult analyze(uint8_t* array, uint64_t count)
{
	AnalysisResult result{ array[0], array[0] };
	for (uint64_t i = 1; i < count; i++)
	{
		uint8_t value = array[i];
		result.min = std::min(result.min, value);
		result.max = std::max(result.max, value);
	}

	return result;
}
