#include "hip/hip_runtime.h"
#include "simulation.h"
#include "timer.h"

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#include <iostream>

__global__ void simulateGravelers(uint64_t seed, uint8_t* results, hiprandState* state, uint64_t simulations)
{
	uint64_t idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < simulations)
	{
		hiprand_init(seed, idx, 0, &state[idx]);
		uint8_t zeroes = 5;
		for (uint8_t i = 0; i < 231; i++) if (hiprand(&state[idx]) % 4 == 0) zeroes++;
		results[idx] = zeroes;
	}
}

uint8_t* simulate(uint64_t seed, uint64_t simulations)
{
	// Allocate cuRAND states
	hiprandState* d_state;
	hipMalloc(&d_state, simulations * sizeof(hiprandState));

	// Allocate results buffer
	uint8_t* d_results;
	hipMalloc(&d_results, simulations);

	// Run simulation kernel
	int threadsPerBlock = 256;
	int blockCount = static_cast<int>(std::ceil(simulations / static_cast<double>(threadsPerBlock)));
	simulateGravelers<<<blockCount, threadsPerBlock>>>(seed, d_results, d_state, simulations);
	
	// Get results
	uint8_t* results = new uint8_t[simulations];
	hipMemcpy(results, d_results, simulations, hipMemcpyDeviceToHost);

	hipFree(d_state);
	hipFree(d_results);
	return results;
}
