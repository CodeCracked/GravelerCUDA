#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <string>
#include <sstream>
#include <cstdlib>

#include "simulation.h"
#include "analysis.h"
#include "timer.h"

uint64_t prompt_uint64(std::string prompt, uint64_t default = 0);
AnalysisResult runSimulations(uint64_t count);

int main()
{
	int seed = time(NULL);
	srand(seed);

	// Prompt Simulation Count
	uint64_t simulations = prompt_uint64("Enter the number of simulations", 100000000);
	uint64_t batchSize = prompt_uint64("Enter the batch size", 100000000);

	// Run Simulation
	__INIT_TIMER__;
	__START_TIMER__;
	AnalysisResult result{};
	uint64_t batchCount = static_cast<uint64_t>(std::ceil(simulations / static_cast<double>(batchSize)));
	uint64_t currentBatch = 1;
	while (simulations > 0)
	{
		uint64_t thisBatchSize = std::min(batchSize, simulations);
		simulations -= thisBatchSize;

		AnalysisResult batchResult = runSimulations(thisBatchSize);
		result.min = std::min(result.min, batchResult.min);
		result.max = std::max(result.max, batchResult.max);
		std::cout << "Finished Batch " << currentBatch++ << " of " << batchCount << std::endl;
	}
	__END_TIMER__
	
	// Print Results
	std::cout << std::endl;
	std::cout << "Minimum Ones Rolled: " << static_cast<int>(result.min) << ", Maximum Ones Rolled: " << static_cast<int>(result.max) << std::endl;
	std::cout << "Time elapsed: " << (elapsedMs / 1000) << " seconds." << std::endl;
	std::cout << "Seed: " << seed << std::endl;

	// Pause
	system("pause");
	return 0;
}

uint64_t prompt_uint64(std::string prompt, uint64_t default)
{
	std::string buffer;
	std::cout << prompt << " (default: " << default << "): ";
	std::getline(std::cin, buffer);

	do
	{
		if (buffer.size() == 0) return default;
		try { return std::stoull(buffer); }
		catch (std::invalid_argument const& ex) {}
		catch (std::out_of_range const& ex) {}
	} while (true);
}
AnalysisResult runSimulations(uint64_t count)
{
	uint8_t* simulationResults = simulate(rand(), count);
	AnalysisResult analysis = analyze(simulationResults, count);
	delete[] simulationResults;
	return analysis;
}